#include <hip/hip_runtime.h>
#include <iostream>

// Vertex data (position and color)
__constant__ float vertexData[] = {
    0.0f,  1.0f, 0.0f, 1.0f,  1.0f, 0.0f, 0.0f, 1.0f,
   -1.0f, -1.0f, 0.0f, 1.0f,  0.0f, 1.0f, 0.0f, 1.0f,
    1.0f, -1.0f, 0.0f, 1.0f,  0.0f, 0.0f, 1.0f, 1.0f
};

// Check if a point is inside a triangle
__device__ bool isInsideTriangle(float px, float py, float* v0, float* v1, float* v2) {
    auto edgeFunction = [](float* a, float* b, float px, float py) {
        return (px - a[0]) * (b[1] - a[1]) - (py - a[1]) * (b[0] - a[0]);
    };
    float w0 = edgeFunction(v1, v2, px, py);
    float w1 = edgeFunction(v2, v0, px, py);
    float w2 = edgeFunction(v0, v1, px, py);
    return (w0 >= 0 && w1 >= 0 && w2 >= 0);
}

// CUDA kernel for rendering
__global__ void renderTriangle(float* frameBuffer, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;

    float px = (x / (float)width) * 2.0f - 1.0f;
    float py = (y / (float)height) * 2.0f - 1.0f;

    float v0[2] = {vertexData[0], vertexData[1]};
    float v1[2] = {vertexData[8], vertexData[9]};
    float v2[2] = {vertexData[16], vertexData[17]};

    if (isInsideTriangle(px, py, v0, v1, v2)) {
        int idx = (y * width + x) * 4; 
        frameBuffer[idx] = 1.0f;     
        frameBuffer[idx + 1] = 0.0f; 
        frameBuffer[idx + 2] = 0.0f; 
        frameBuffer[idx + 3] = 1.0f; 
    }
}

extern "C" void generateTriangle(float* frameBuffer, int width, int height) {
    float* d_frameBuffer;
    const size_t frameBufferSize = width * height * 4 * sizeof(float);

    hipMalloc(&d_frameBuffer, frameBufferSize);
    hipMemset(d_frameBuffer, 0, frameBufferSize);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    renderTriangle<<<gridSize, blockSize>>>(d_frameBuffer, width, height);
    hipMemcpy(frameBuffer, d_frameBuffer, frameBufferSize, hipMemcpyDeviceToHost);

    hipFree(d_frameBuffer);
}

